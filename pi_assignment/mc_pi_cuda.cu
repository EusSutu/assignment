/////////////////////////////////////////////////////////////////////////
//                                                                     //
//  CUDA code which calculates PI using Monte-Carlo method             //
//  It will get random points in the square between (0,0) and (1,1)    //
//  Find whether it is in the circle of radius 1 and use it to find PI //
//  Name: Eusebiu Sutu                                                 //
//  E-mail: eusebiu.sutu@lincoln.ox.ac.uk                              //
//  Date: May 25th, 2018                                               //
//  CWM: High performance computing                                    //
//                                                                     //
/////////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.1415926536

//Kernel which verifies whether points are in the circle
//If it is then add to area

__global__ void mc_pi (int *area_d,float *f_rand_d,int N)
{
  float r2; //Distance squared from point to origin
  int index = 2*(blockIdx.x*N+threadIdx.x);
  float x,y;

  //Coordinates are stored as {point0.x,point0.y,point1.x,point1.y,...}
  x = f_rand_d[index];
  y = f_rand_d[index+1];

  r2 = x*x + y*y;
  if(r2 <= 1)
    atomicAdd(area_d, 1); 
}

//Function which inputs different values to the number of points and calculates pi and th error

void error(hiprandGenerator_t gen)
{
   float *f_rand_d,pi;  //f_rand_d are the coordinates generated, pi will be the value for PI calculated  
   int i;               //i*i is the total number of points generated
   int area,*area_d;    //area_d is the area calculated on the device, area is a host copy of it

   for (i=2;i<=1024;i*=2)
   {
     area = 0;          // reset
     hipMalloc( (void **) &f_rand_d, 2*i*i*sizeof(float));           // allocate device memory
     hipMalloc( (void **) &area_d, sizeof(int));

     hipMemcpy(area_d, &area, sizeof(int), hipMemcpyHostToDevice);  // reset area_d

     hiprandGenerateUniform( gen, f_rand_d, 2*i*i);                    //generate the coordinates

     mc_pi<<<i,i>>>(area_d,f_rand_d,i);                               //calculate area

     hipMemcpy(&area, area_d, sizeof(int), hipMemcpyDeviceToHost);  //get result back to host

     pi = (float)area/((float)i*(float)i);                            //calculate pi from area
     pi*=4;

     printf("i=%d  pi=%f error=%f\n",i,pi,pi-PI);                     //print results out on the console

     hipFree(f_rand_d);                                              //free memory
     hipFree(area_d);
   }
}

int main()
{
  //Initialize the GPU
  
  int deviceid=0;
  int devCount;
  hipGetDeviceCount(&devCount);
  if(deviceid<devCount) hipSetDevice(deviceid);
  else return 1;
  

  //Create the generator
  hiprandGenerator_t gen;  
  hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL);

  //Use the error funtion
  error(gen);

  hipDeviceReset();
}
